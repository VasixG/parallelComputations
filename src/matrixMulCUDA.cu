#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void matrixMulCUDA(float *C, float *A, float *B, int width) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if(row < width && col < width) {
        float sum = 0;
        for (int k = 0; k < width; k++) {
            sum += A[row * width + k] * B[k * width + col];
        }
        C[row * width + col] = sum;
    }
}

int main() {

    FILE *fileCUDAfull = fopen("timesCUDAfull.txt", "w");
    if (fileCUDAfull == NULL) {
        printf("Error opening file!\n");
        return 1;
    }

    FILE *fileCUDA = fopen("timesCUDA.txt", "w");
    if (fileCUDA == NULL) {
        printf("Error opening file!\n");
        return 1;
    }

    int startN = 1;
    int stepN = 1;
    int maxN = 500;

    
    int numOfIterations = (maxN / stepN);
    double timesCUDA[numOfIterations];
    double timesCUDAfull[numOfIterations];

    for(int N = startN, id =0; N < maxN; N+=stepN, ++id){
        float milliseconds = 0;
        double full_time_used = 0;
        clock_t startf, endf;

        startf = clock();
        hipEvent_t start, stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

        size_t size = N * N * sizeof(float);
        float *A, *B, *C; 
        float *d_A, *d_B, *d_C; 

        hipMalloc((void **)&d_A, size);
        hipMalloc((void **)&d_B, size);
        hipMalloc((void **)&d_C, size);

        A = (float *)malloc(size);
        B = (float *)malloc(size);
        C = (float *)malloc(size);

        for (int i = 0; i < N; i++) {
            for (int j = 0; j < N; j++) {
                A[i * N + j] = sin(i) * cos(j); 
            }
        }

        for (int i = 0; i < N; i++) {
            for (int j = 0; j < N; j++) {
                B[i * N + j] = cos(i) * sin(j); 
            }
        }

        hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

        dim3 threadsPerBlock(16, 16);
        dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                          (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
        matrixMulCUDA<<<blocksPerGrid, threadsPerBlock>>>(d_C, d_A, d_B, N);

        hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&milliseconds, start, stop);

        timesCUDA[id] = milliseconds/1000;
        endf = clock();
        full_time_used = ((double) (endf - startf)) / CLOCKS_PER_SEC;

        timesCUDAfull[id] = full_time_used;
        hipEventDestroy(start);
        hipEventDestroy(stop);

        hipFree(d_A); hipFree(d_B); hipFree(d_C);
        free(A); free(B); free(C);
    } 

    for (int i = 0; i < numOfIterations; i++) {
        fprintf(fileCUDAfull, "%d: %f\n", (i + 1) * stepN, timesCUDAfull[i]);
    }

    for (int i = 0; i < numOfIterations; i++) {
        fprintf(fileCUDA, "%d: %f\n", (i + 1) * stepN, timesCUDA[i]);
    }

    fclose(fileCUDA);
    fclose(fileCUDAfull);
    

    return 0;
}
